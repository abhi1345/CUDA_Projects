#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <math.h>

int nextPowerOf2(int size) {
    if (size == 0) return 1;
    return pow(2, ceil(log2(size)));
}

// CUDA Error Check Macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void BitonicSortCUDA(int* arrGpu, int size, int i, int j) {
    // Shared memory for faster access
    // extern __shared__ int sharedMem[];

    // Calculate thread index and map it to array index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // // Copy data to shared memory (each thread loads one element)
    // if (idx < size) {
    //     sharedMem[threadIdx.x] = arrGpu[idx];
    // }
    // __syncthreads();

    // Perform the bitonic sort
    int two_pow_i = 1 << i;
    int two_pow_j = 1 << j;

    // Calculate indices based on thread index
    int first_index = idx; //threadIdx.x;
    int second_index = first_index ^ two_pow_j;

    if (second_index > first_index && second_index < size) {
        int first_val = arrGpu[first_index];
        int second_val = arrGpu[second_index];

        // Sorting condition based on bitonic pattern
        if ((two_pow_i & first_index) == 0) {
            if (first_val > second_val) {
                // Swap if out of order
                arrGpu[first_index] = second_val;
                arrGpu[second_index] = first_val;
            }
        } else {
            if (first_val < second_val) {
                // Swap if out of order
                arrGpu[first_index] = second_val;
                arrGpu[second_index] = first_val;
            }
        }
    }

    // Copy sorted data back to global memory
    // if (idx < size) {
    //     arrGpu[idx] = sharedMem[threadIdx.x];
    // }
    __syncthreads();
}


int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Usage: %s <array_size>\n", argv[0]);
        return 1;
    }

    int size = atoi(argv[1]);

    srand(time(NULL));

    // ======================================================================
    // arCpu contains the input random array
    // arrSortedGpu should contain the sorted array copied from GPU to CPU
    // ======================================================================
    int* arrCpu = (int*)malloc(size * sizeof(int));
    int* arrSortedGpu = (int*)malloc(size * sizeof(int));

    for (int i = 0; i < size; i++) {
        arrCpu[i] = rand() % 1000;
    }

    // Print the original array
    // printf("Original Array:\n");
    // for (int i = 0; i < size; i++) {
    //     printf("%d ", arrCpu[i]);
    // }
    // printf("\n");

    float gpuTime, h2dTime, d2hTime, cpuTime = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // ======================================================================
    // Transfer data (arr_cpu) to device
    // ======================================================================

    int *arrGpu;
    hipMallocManaged(&arrGpu, size * sizeof(int));
    hipMemcpy(arrGpu, arrCpu, size * sizeof(int), hipMemcpyHostToDevice);

    // ======================================================================
    // End your code
    // ======================================================================

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&h2dTime, start, stop);

    hipEventRecord(start);
    
    // ======================================================================
    // Perform bitonic sort on GPU
    // ======================================================================

    int threadsPerBlock = 1024;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    // size_t sharedMemSize = threadsPerBlock * sizeof(int);

    // Launch the kernel
    // BitonicSortCUDA<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(arrGpu, size);

    int log_len = 31 - __builtin_clz(size);

    for (int i = 1; i <= log_len; i++) 
    {
        for (int j = i-1; j >= 0; j--) 
        {
            BitonicSortCUDA<<<blocksPerGrid, threadsPerBlock>>>(arrGpu, size, i, j);
        }
    }

    // BitonicSortCUDA<<<blocksPerGrid, threadsPerBlock>>>(arrGpu, size);


    // ======================================================================
    // End your code
    // ======================================================================

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    hipEventRecord(start);

    // ======================================================================
    // Transfer sorted data back to host (copied to arr_sorted_gpu)
    // ======================================================================

    hipMemcpy(arrSortedGpu, arrGpu, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(arrGpu);

    // ======================================================================
    // End your code
    // ======================================================================

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&d2hTime, start, stop);

    // Print the sorted array (from GPU)
    // printf("Sorted Array (GPU):\n");
    // for (int i = 0; i < size; i++) {
    //     printf("%d ", arrSortedGpu[i]);
    // }
    // printf("\n");


    auto startTime = std::chrono::high_resolution_clock::now();
    
    // CPU sort for performance comparison
    std::sort(arrCpu, arrCpu + size);
    
    auto endTime = std::chrono::high_resolution_clock::now();
    cpuTime = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();
    cpuTime = cpuTime / 1000;

    int match = 1;
    for (int i = 0; i < size; i++) {
        if (arrSortedGpu[i] != arrCpu[i]) {
            match = 0;
            break;
        }
    }

    free(arrCpu);
    free(arrSortedGpu);

    if (match)
        printf("\033[1;32mFUNCTIONAL SUCCESS\n\033[0m");
    else {
        printf("\033[1;31mFUNCTIONCAL FAIL\n\033[0m");
        return 0;
    }
    
    printf("\033[1;34mArray size         :\033[0m %d\n", size);
    printf("\033[1;34mCPU Sort Time (ms) :\033[0m %f\n", cpuTime);
    float gpuTotalTime = h2dTime + gpuTime + d2hTime;
    int speedup = (gpuTotalTime > cpuTime) ? (gpuTotalTime/cpuTime) : (cpuTime/gpuTotalTime);
    float meps = size / (gpuTotalTime * 0.001) / 1e6;
    printf("\033[1;34mGPU Sort Time (ms) :\033[0m %f\n", gpuTotalTime);
    printf("\033[1;34mGPU Sort Speed     :\033[0m %f million elements per second\n", meps);
    if (gpuTotalTime < cpuTime) {
        printf("\033[1;32mPERF PASSING\n\033[0m");
        printf("\033[1;34mGPU Sort is \033[1;32m %dx \033[1;34mfaster than CPU !!!\033[0m\n", speedup);
        printf("\033[1;34mH2D Transfer Time (ms):\033[0m %f\n", h2dTime);
        printf("\033[1;34mKernel Time (ms)      :\033[0m %f\n", gpuTime);
        printf("\033[1;34mD2H Transfer Time (ms):\033[0m %f\n", d2hTime);
    } else {
        printf("\033[1;31mPERF FAILING\n\033[0m");
        printf("\033[1;34mGPU Sort is \033[1;31m%dx \033[1;34mslower than CPU, optimize further!\n", speedup);
        printf("\033[1;34mH2D Transfer Time (ms):\033[0m %f\n", h2dTime);
        printf("\033[1;34mKernel Time (ms)      :\033[0m %f\n", gpuTime);
        printf("\033[1;34mD2H Transfer Time (ms):\033[0m %f\n", d2hTime);
        return 0;
    }

    return 0;
}

