#define TILE_WIDTH 4

// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <cstring> // Added for strcmp
#include <ctime>
#include <chrono>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Function to compare result matrices from CPU and GPU
bool compareMatrices(float* C, float* D, int size) 
{
    for (int i = 0; i < size; ++i) {
        float error = std::abs(C[i] - D[i]);
        if (error > 1e-3) {
            return false;
        }
    }
    return true;
}

// Function to initialize matrices A and B
void initializeMatrices(float* matrix, int size) 
{
    std::srand(static_cast<unsigned>(std::time(nullptr)));
    for (int i = 0; i < size; ++i) 
    {
        // Generate a random float number between 0 and 1
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// CPU implementation
void matrixMultiplication(float* A, float* B, float* D, int w) 
{
    for (int i = 0; i < w; ++i) {
        for (int j = 0; j < w; ++j) {
            float sum = 0.0;
            for (int k = 0; k < w; ++k) {
                sum += A[i * w + k] * B[k * w + j];
            }
            D[i * w + j] = sum;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
// Do not modify code above this line
////////////////////////////////////////////////////////////////////////////////////////////////////////

// GPU implementation
// Implement this kernel function
// A & B are addresses on the host for input matrices, C is the address on the host for output matrix
// matrixWidth is the width of matrices for which matrix multiplication is being performed
__global__ void MatrixMulCUDA(float* C, float* A, float* B, int matrixWidth) 
{
    __shared__ float Ashared[TILE_WIDTH][16];
    __shared__ float Bshared[16][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int gridSize = matrixWidth / TILE_WIDTH;

    int blockRow = by * TILE_WIDTH; // Starting corner for this block
    int blockCol = bx * TILE_WIDTH; // Starting corner for this block

    for (int i = 0; i < TILE_WIDTH; i++) 
    {
        for (int j = 0; j < matrixWidth; j++) 
        {
            Ashared[i][j] = A[(blockRow+i)*matrixWidth + j];
            Bshared[j][i] = B[j*matrixWidth + blockCol+i];
        }
    }

    for (int i = 0; i < TILE_WIDTH; i++)
    {
        for (int j = 0; j < TILE_WIDTH; j++)
        {
            float curSum = 0;
            for (int k = 0; k < matrixWidth; k++)
            {
                curSum += Ashared[i][k] * Bshared[k][j];
            }
            C[(blockRow+i)*matrixWidth + blockCol+j] = curSum;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
// Do not modify code below this line
////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
 * Program main
 */
int main(int argc, char** argv) 
{
    if (argc != 3 || strcmp(argv[1], "-m") != 0) 
    {
        std::cout << "Usage: ./a.out -m <matrix width>" << std::endl;
        return -1;
    }

    int matrixWidth = atoi(argv[2]);
    
    int matrixSize = matrixWidth * matrixWidth;

    float *A, *B, *C, *D;
    hipMallocManaged(&A, matrixSize * sizeof(float));
    hipMallocManaged(&B, matrixSize * sizeof(float));
    hipMallocManaged(&C, matrixSize * sizeof(float));
    hipMallocManaged(&D, matrixSize * sizeof(float));

    initializeMatrices(A, matrixSize);
    initializeMatrices(B, matrixSize);

    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize(matrixWidth / TILE_WIDTH, matrixWidth / TILE_WIDTH);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpu_time = 0;
    float cpu_time = 0;

    hipEventRecord(start);
    // Launch the kernel
    MatrixMulCUDA<<<gridSize, blockSize>>>(C, A, B, matrixWidth);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&gpu_time, start, stop);

    auto start_time = std::chrono::high_resolution_clock::now();
    // Perform matrix multiplication on CPU and store in D
    matrixMultiplication (A, B, D, matrixWidth);    
    auto end_time = std::chrono::high_resolution_clock::now();
    cpu_time = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count();
    cpu_time = cpu_time / 1000;

    // ... (Perform matrix multiplication on CPU and store in D)

    // Compare matrices C and D
    bool matricesMatch = compareMatrices(C, D, matrixSize);
    
    if (matricesMatch) {
        printf("SUCCESS!\n");
        printf("CPU Matrix Multiply Time (ms) : %f \n", cpu_time);
        printf("GPU Matrix Multiply Time (ms) : %f \n", gpu_time);
	printf("Speedup: %f \n", cpu_time/gpu_time);
    } else {
        printf("ERROR!\n");
    }

    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(D);

    return 0;    
}
